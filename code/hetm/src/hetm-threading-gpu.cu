#include "hip/hip_runtime.h"
#include "hetm-log.h"
#include "hetm.cuh"
#include "pr-stm-wrapper.cuh"
#include "hetm-timer.h"
#include "hetm-cmp-kernels.cuh"
#include "knlman.hpp"
#include "arch.h"

#include "graph.h"
// #include "gsort_lib.h"
#include "bb_lib.h"

#include <list>

using namespace memman;

#define MINIMUM_DtD_CPY 4194304
#define MINIMUM_DtH_CPY 262144
#define MINIMUM_HtD_CPY 262144

pr_tx_args_s HeTM_pr_args[HETM_NB_DEVICES]; // used only by the CUDA-control thread (TODO: PR-STM only)

int isAfterCmpDone = 0;
int isGetStatsDone = 0;
int isGetPRStatsDone = 0;
int isDatasetSyncDone = 0;
int isGPUResetDone = 0;
long roundCountAfterBatch = 0;
int GPU_merger_id = 0;

static int isGPUDetectInterConfl = 0; // used in the main-loop --> TODO: get rid of this

static std::list<HeTM_callback> beforeGPU;
static std::list<HeTM_callback> afterGPU;
static std::list<HeTM_callback> beforeBatch;
static std::list<HeTM_callback> afterBatch;
static std::list<HeTM_callback> beforeKernel;
static std::list<HeTM_callback> afterKernel;
static HeTM_callback choose_policy;

static long lastRoundTXs[HETM_NB_DEVICES+1];

// static inline void waitDatasetEnd();
static void startGPUtoGPUconflictCompare(int id);
static void waitGPUtoGPUconflictCompare();
static int launchInterGPUConflDetectKernel(HeTM_thread_s *threadData, int locGPU, int remGPU);

// executed in other thread
// static void offloadResetGPUState(void*);

// static thread_local HeTM_thread_s *tmp_threadData; // TODO: HETM_OVERLAP_CPY_BACK

static int peerCpyAvailable[HETM_NB_DEVICES*HETM_NB_DEVICES];

void initGPUPeerCpy()
{
  int nGPUs = Config::GetInstance()->NbGPUs();
  int nbOfGPUs = Config::GetInstance()->GetNbPhysicalGPUs();
  CUDA_CHECK_ERROR(hipGetDeviceCount(&nbOfGPUs), "");
  for (int j = 0; j < nGPUs; ++j) {
    peerCpyAvailable[j*nGPUs + j] = 1;
    for (int k = j+1; k < nGPUs; ++k) {
      int coord1 = j*nGPUs + k, coord2 = k*nGPUs + j;
      int coord1Real = (j%nbOfGPUs)*nGPUs + (k%nbOfGPUs);
      int coord2Real = (k%nbOfGPUs)*nGPUs + (j%nbOfGPUs);
      if (coord1 == coord1Real && coord2 == coord2Real) {
        CUDA_CHECK_ERROR(hipDeviceCanAccessPeer(&(peerCpyAvailable[coord1]), j, k), "");
        CUDA_CHECK_ERROR(hipDeviceCanAccessPeer(&(peerCpyAvailable[coord2]), k, j), "");
        if (peerCpyAvailable[coord1]) {
          CUDA_CHECK_ERROR(hipSetDevice(j), "");
          CUDA_CHECK_ERROR(hipDeviceEnablePeerAccess(k, 0), "");
        }
        if (peerCpyAvailable[coord2]) {
          CUDA_CHECK_ERROR(hipSetDevice(k), "");
          CUDA_CHECK_ERROR(hipDeviceEnablePeerAccess(j, 0), "");
        }
      } else {
        peerCpyAvailable[coord1] = peerCpyAvailable[coord1Real];
        peerCpyAvailable[coord2] = peerCpyAvailable[coord2Real];
      }
    }
  }
}

void destroyGPUPeerCpy()
{
  int nGPUs = Config::GetInstance()->NbGPUs();
  int nbOfGPUs = Config::GetInstance()->GetNbPhysicalGPUs();
  
  for (int j = 0; j < nGPUs; ++j)
  {
    peerCpyAvailable[j*nGPUs + j] = 1;
    for (int k = j+1; k < nGPUs; ++k)
    {
      int coord1 = j*nGPUs + k, coord2 = k*nGPUs + j;
      int coord1Real = (j%nbOfGPUs)*nGPUs + (k%nbOfGPUs);
      int coord2Real = (k%nbOfGPUs)*nGPUs + (j%nbOfGPUs);
      if (coord1 == coord1Real && coord2 == coord2Real)
      {
        if (peerCpyAvailable[coord1])
        {
          CUDA_CHECK_ERROR(hipSetDevice(j), "");
          CUDA_CHECK_ERROR(hipDeviceDisablePeerAccess(k), "");
        }
        if (peerCpyAvailable[coord2])
        {
          CUDA_CHECK_ERROR(hipSetDevice(k), "");
          CUDA_CHECK_ERROR(hipDeviceDisablePeerAccess(j), "");
        }
      }
    }
  }
}

struct runMultiGPUbatch_
{
  int threadId;
  int devId;
  HeTM_callback callback;
  void *clbkArgs;
};

// TODO: use multiple threads to launch the kernels
static void runMultiGPUbatch(struct runMultiGPUbatch_ *a)
{
  // printf("         <<<<<<<<<< runMultiGPUbatch >>>>>>>>>>\n");
  Config::GetInstance()->SelDev(a->devId);
  a->callback(a->threadId, a->clbkArgs);
}

static struct runMultiGPUbatch_ runMultiGPUbatch_args[HETM_NB_DEVICES];

void runGPUBatch()
{
  // printf("                 <<<<<<<<<< runGPUBatch >>>>>>>>>>\n");
  int threadId = HeTM_thread_data[0]->id;
  HeTM_callback callback = HeTM_thread_data[0]->callback;
  void *clbkArgs = HeTM_thread_data[0]->args;
  if (HeTM_get_GPU_status(0) != HETM_IS_EXIT) {
    for (int j = 0; j < Config::GetInstance()->NbGPUs(); ++j)
    {
      // while (__atomic_load_n(&HeTM_gpu_has_work[j], __ATOMIC_ACQUIRE)
      //   && !HeTM_async_is_stop(0)); // previous work

      runMultiGPUbatch_args[j].threadId = threadId;
      runMultiGPUbatch_args[j].devId = j;
      runMultiGPUbatch_args[j].callback = callback;
      runMultiGPUbatch_args[j].clbkArgs = clbkArgs;

      // TODO: multi thread is not working
      runMultiGPUbatch(&runMultiGPUbatch_args[j]);
    }
  }

  // for (int j = 0; j < HETM_NB_DEVICES; ++j)
  // {
  //   while (__atomic_load_n(&HeTM_gpu_has_work[j], __ATOMIC_ACQUIRE)
  //     && !HeTM_async_is_stop(0));
  // }
}

void runBeforeGPU(int id, void *data)
{
  HeTM_gshared_data.batchCount = 1; // TODO: first batch, this value is used to mark the bitmaps
  for (auto it = beforeGPU.begin(); it != beforeGPU.end(); ++it) {
    HeTM_callback clbk = *it;
    clbk(id, data);
  }
}

void runAfterGPU(int id, void *data)
{
  for (auto it = afterGPU.begin(); it != afterGPU.end(); ++it) {
    HeTM_callback clbk = *it;
    clbk(id, data);
  }
}

void runGPUBeforeBatch(int id, void *data)
{
  for (int devId = 0; devId < Config::GetInstance()->NbGPUs(); ++devId) {
    for (auto it = beforeBatch.begin(); it != beforeBatch.end(); ++it) {
      HeTM_callback clbk = *it;
      clbk(id, data);
    }
  }
}

void runGPUAfterBatch(int id, void *data)
{
  HeTM_gshared_data.batchCount++;
  if ((HeTM_gshared_data.batchCount & 0xff) == 0) {
    HeTM_gshared_data.batchCount++;
  }
  __sync_synchronize();
  for (auto it = afterBatch.begin(); it != afterBatch.end(); ++it) {
    HeTM_callback clbk = *it;
    clbk(id, data);
  }
}

void runGPUBeforeKernel(int id, void *data)
{
  for (auto it = beforeKernel.begin(); it != beforeKernel.end(); ++it) {
    HeTM_callback clbk = *it;
    clbk(id, data);
  }
}

void runGPUAfterKernel(int id, void *data)
{
  for (auto it = afterKernel.begin(); it != afterKernel.end(); ++it)
  {
    HeTM_callback clbk = *it;
    clbk(id, data);
  }

  for (int j = 0; j < Config::GetInstance()->NbGPUs(); ++j)
  {
    Config::GetInstance()->SelDev(j);
    PR_curr_dev = j;
    __atomic_or_fetch(&isGPUDetectInterConfl, HeTM_is_interconflict(j), __ATOMIC_RELEASE);
  }
}

pr_tx_args_s *getPrSTMmetaData(int devId)
{
  return &(HeTM_pr_args[devId]);
}

void waitGPUBatchEnd()
{
  int j;
  for (j = 0; j < Config::GetInstance()->NbGPUs(); ++j)
  {
    Config::GetInstance()->SelDev(j);
    PR_curr_dev = j;
    // auto strm = PR_getCurrentStream();
    if (!HeTM_async_is_stop(0))
    {
      PR_waitKernel(&HeTM_pr_args[PR_curr_dev]);
      PR_useNextStream(&HeTM_pr_args[PR_curr_dev]);
    }
    // CUDA_CHECK_ERROR(hipStreamSynchronize(strm), "");
  }
  // Removed deviceSync from here
}

void notifyBatchIsDone()
{
  int i, j;
  int nGPUs = Config::GetInstance()->NbGPUs();
  for (j = 0; j < nGPUs; j++)
  {
    // size_t cpyrdsetBMAPcache = 0;
    size_t cpywrsetBMAPcache = 0;
    Config::GetInstance()->SelDev(j);
    MemObj *m = HeTM_gpu_wset_cache.GetMemObj(j);
    m->CpyDtH(HeTM_memStream2[j]);
    cpywrsetBMAPcache += m->size;
    
    __atomic_add_fetch(&HeTM_stats_data.sizeCpyWSetHtD, cpywrsetBMAPcache, __ATOMIC_ACQ_REL);
  }

  // wait the previous copies
  for (j = 0; j < nGPUs; j++)
    CUDA_CHECK_ERROR(hipStreamSynchronize((hipStream_t)HeTM_memStream2[j]), "");
  
  for (j = 0; j < nGPUs; j++)
  {
    HeTM_set_GPU_status(j, HETM_BATCH_DONE); // notifies
    if (HeTM_gshared_data.isCPUEnabled)
      for (i = 0; i < nGPUs; ++i)
        startGPUtoGPUconflictCompare(i);
  }
  __sync_synchronize();
  if (HeTM_gshared_data.isCPUEnabled)
    waitGPUtoGPUconflictCompare();
}

void waitCPUlogValidation(int nonBlock)
{
  int j;
  // waits threads to stop doing validation (VERS)
  // if (!nonBlock)
  //   HeTM_sync_next_batch(); // wait CPU side comparisons
  for (j = 0; j < Config::GetInstance()->NbGPUs(); j++)
  {
    Config::GetInstance()->SelDev(j);
    HETM_DEB_THRD_GPU("GPU %i waits CPU log validation", j);
    do {
      COMPILER_FENCE();
      CUDA_CHECK_ERROR(hipDeviceSynchronize(), "waiting CPU validation");
    } while ( !nonBlock &&
      __atomic_load_n(&(HeTM_shared_data[j].threadsWaitingSync), __ATOMIC_ACQUIRE)
      < HeTM_gshared_data.nbCPUThreads && !HeTM_is_stop(0)
    );
  }
}

void syncGPUtoCPUbarrier(int nonBlock)
{
  int j;
  // waits threads to stop doing validation (VERS)

  // TODO: isInterGPUConflDone was waited and set before by the GPU in notifyBatchIsDone
  for (j = 0; j < Config::GetInstance()->NbGPUs(); j++)
  {
    if (!nonBlock)
      HeTM_sync_barrier(j);
  }
}


void waitGPUCMPEnd(int nonBlock)
{
  waitCPUlogValidation(nonBlock);
  syncGPUtoCPUbarrier(nonBlock);
}

void mergeGPUDataset()
{
  // TODO: this function is not merging anything
  RUN_ASYNC(getGPUStatistics, NULL);

  checkIsExit();
  // ---------------------
  // TIMER_READ(t1WCpy);
  RUN_ASYNC(syncGPUdataset, HeTM_thread_data[0]);
  RUN_ASYNC(waitGPUdataset, HeTM_thread_data[0]);
}

void doGPUStateReset()
{
  RUN_ASYNC(offloadResetGPUState, NULL);
}

int HeTM_choose_policy(HeTM_callback req)
{
  choose_policy = req;
  return 0;
}

int HeTM_before_gpu_start(HeTM_callback req)
{
  beforeGPU.push_back(req);
  return 0;
}

int HeTM_after_gpu_finish(HeTM_callback req)
{
  afterGPU.push_back(req);
  return 0;
}

int HeTM_before_batch(HeTM_callback req)
{
  beforeBatch.push_back(req);
  return 0;
}

int HeTM_after_batch(HeTM_callback req)
{
  afterBatch.push_back(req);
  return 0;
}

int HeTM_before_kernel(HeTM_callback req)
{
  beforeKernel.push_back(req);
  return 0;
}

int HeTM_after_kernel(HeTM_callback req)
{
  afterKernel.push_back(req);
  return 0;
}

void hetm_memcpyDeviceToCPU(int devId, HeTM_thread_s *threadData)
{
  size_t datasetCpySize = 0;
  void *devWinner;

  if (devId < Config::GetInstance()->NbGPUs())
  { // some GPU won
    Config::GetInstance()->SelDev(devId);
    MemObj *m = HeTM_mempool.GetMemObj(devId);
    devWinner = m->dev;
    auto strm = PR_getCurrentStream();
    void *host =  m->host;
    datasetCpySize = m->size;

    // copy to CPU
    CUDA_CHECK_ERROR(
      hipMemcpyAsync(host, devWinner, datasetCpySize, hipMemcpyDeviceToHost, strm),
      "");

    HeTM_stats_data.sizeCpyDataset += datasetCpySize;
  }
}

static void mergeAllMatrices()
{
  int nGPUs = Config::GetInstance()->NbGPUs();

  for (int j = 0; j < (nGPUs+1); ++j)
  {
    for (int i = 0; i < nGPUs; ++i)
    {
      // int coord_c = (nGPUs+1)*j + i; // column
      int coord_l = (nGPUs+1)*i + j; // row
      unsigned char GPUval = __atomic_load_n(&(HeTM_shared_data[i].mat_confl_GPU_unif[coord_l]), __ATOMIC_ACQUIRE);
      HeTM_gshared_data.mat_confl_CPU_unif[coord_l] |= GPUval;
      if (j == nGPUs)
      { // CPU does not do validation, need to add the mirrored conflict
        int coord_c = (nGPUs+1)*j + i; // column
        HeTM_gshared_data.mat_confl_CPU_unif[coord_c] |= GPUval;
      }
      // printf("mergeAllMatrices[%i][%i] = %i\n", i, j, (int)HeTM_shared_data[0].mat_confl_CPU_unif[coord_l]);
    }
  }
}

#ifdef HETM_DEB
static void printMatrixPerGPU()
{
  int nGPUs = Config::GetInstance()->NbGPUs();

  HETM_DEB_THRD_GPU("\n Conflict matrices:");
  for (int j = 0; j < nGPUs+1; ++j) {
    if (j == nGPUs) {
      printf("       >>> CPU  \n   \\ ");
    }
    else {
      printf("       >>> GPU %i \n   \\ ", j);
    }
    for (int k = 0; k < nGPUs+1; ++k) {
      if (k == nGPUs) {
        printf(" CPU ");
      } 
      else {
        printf(" GPU%i", k);
      }
    }
    for (int k = 0; k < nGPUs+1; ++k) {
      if (k == nGPUs) { printf("\n CPU "); }
      else { printf("\n GPU%i", k); }
      if (j == nGPUs)
      {
        for (int l = 0; l < nGPUs+1; ++l) {
          int coord1 = (nGPUs+1)*k + l;
          printf("   %i ", (int)HeTM_gshared_data.mat_confl_CPU_unif[coord1]);
        }
      }
      else 
      {
        for (int l = 0; l < nGPUs+1; ++l) {
          int coord1 = (nGPUs+1)*k + l;
          printf("   %i ", (int)HeTM_shared_data[j].mat_confl_GPU_unif[coord1]);
        }
      }
    }
    printf("\n");
  }
  printf("\n");
}
#endif

static void
startGPUtoGPUconflictCompare(
  int devId
) {
  // this is done serially
  static unsigned char lastBatch = 0; // TODO: remove lastBatch
  static int countDevs = 0;
  unsigned char batch = (unsigned char) HeTM_gshared_data.batchCount;
  int nGPUs = Config::GetInstance()->NbGPUs();

  HETM_DEB_THRD_GPU("\033[0;31m" "Thread %i batch(%i) != lastBatch(%i)" "\033[0m",
    devId, batch, lastBatch);

  if (batch != lastBatch) {
    size_t cpywrsetBMAP = 0;

    if (!(nGPUs > 1)) // only 1 GPU does not do inter-GPU conflict detection
      __atomic_store_n(&(HeTM_shared_data[0].isInterGPUConflDone), 1, __ATOMIC_RELEASE);

    for (int j = devId+1; j < nGPUs; ++j)
    {
      auto threadDataj = &(HeTM_shared_data[j].threadsInfo[0]);
      auto threadDatak = &(HeTM_shared_data[devId].threadsInfo[0]);

      int devSrc = devId;
      int devDst = j;

      MemObj *m_dst_remote_wset = HeTM_gpu_wset_ext[devDst*nGPUs+devSrc].GetMemObj(devDst);
      MemObj *m_src_wset = HeTM_gpu_wset.GetMemObj(devSrc);
      MemObj *m_src_wset_cache = HeTM_gpu_wset_cache.GetMemObj(devSrc);
      MemObj *m_src_wset_filter = m_src_wset;

      MemObjCpyBuilder b;
      MemObjCpyDtD m(b // know how to do PtP also
        .SetGranFilter(8) // TODO
        ->SetGranApply(sizeof(char))
        ->SetForceFilter(0)
        ->SetFilterVal(HeTM_gshared_data.batchCount)
        ->SetCache(m_src_wset_cache)
        ->SetFilter(m_src_wset_filter)
        ->SetDst(m_dst_remote_wset)
        ->SetSrc(m_src_wset)
        ->SetSizeChunk(BMAP_GRAN)
        ->SetStrm1(HeTM_memStream[devDst])
        ->SetStrm2(HeTM_memStream2[devDst])
      );
      cpywrsetBMAP = m.Cpy();


      // int dev0 = GetActualDev(devId);
      // int dev1 = GetActualDev(j);

      // MemObj *m0 = HeTM_gpu_wset_cache.GetMemObj(devId);
      // MemObj *m1 = HeTM_gpu_wset_cache.GetMemObj(j);

      // uintptr_t src0, src1;
      // uintptr_t dst0, dst1;
      // long gran = BMAP_GRAN;
      // long sz = gran;
      // long cacheSize;
      // size_t size = HeTM_gshared_data.sizeMemPool/sizeof(PR_GRANULE_T);
      // cacheSize = (size + (gran-1)) / gran;
      // hipStream_t strm0 = (hipStream_t)HeTM_memStream[j];
      // hipStream_t strm1 = (hipStream_t)HeTM_memStream[devId];

      // // char *GPUwsPtr_devptr0 = (char*)HeTM_shared_data[devId].bmap_wset_GPU_devptr;
      // char *GPUwsPtrCache_hostptr0 = (char*)m0->host;
      
      // // char *GPUwsPtr_devptr1 = (char*)HeTM_shared_data[j].bmap_wset_GPU_devptr;
      // char *GPUwsPtrCache_hostptr1 = (char*)m1->host;

      // dst0 = (uintptr_t)HeTM_shared_data[j].bmap_wset_GPU_devptr[devId];
      // src0 = (uintptr_t)HeTM_shared_data[devId].bmap_wset_GPU_devptr[devId];
      // dst1 = (uintptr_t)HeTM_shared_data[devId].bmap_wset_GPU_devptr[j];
      // src1 = (uintptr_t)HeTM_shared_data[j].bmap_wset_GPU_devptr[j];

      // for (int i = 0; i < cacheSize; ++i)
      // {
      //   // last one may need to redefine the size
      //   if (i == cacheSize-1 || size < gran)
      //     sz = size - (gran * (cacheSize-1));

      //   if (GPUwsPtrCache_hostptr0[i] == batch)
      //   {
      //     cpywrsetBMAP += sz;
      //     if (dev0 == dev1)
      //     {
      //       CUDA_CPY_DtD_ASYNC(
      //         dst0, src0, sz, strm0
      //       );
      //     }
      //     else
      //     {
      //       CUDA_CPY_PtP_ASYNC(
      //         dst0, dev1, src0, dev0, sz, strm0
      //       );
      //     }
      //     strm0 = strm0 == HeTM_memStream[j] ? (hipStream_t)HeTM_memStream2[j] : (hipStream_t)HeTM_memStream[j];
      //   }
      //   if (GPUwsPtrCache_hostptr1[i] == batch)
      //   {
      //     cpywrsetBMAP += sz;
      //     if (dev0 == dev1)
      //     {
      //       CUDA_CPY_DtD_ASYNC(
      //         dst1, src1, sz, strm1
      //       );
      //     }
      //     else
      //     {
      //       CUDA_CPY_PtP_ASYNC(
      //         dst1, dev0, src1, dev1, sz, strm1
      //       );
      //     }
      //     strm1 = strm1 == HeTM_memStream[devId] ? (hipStream_t)HeTM_memStream2[devId] : (hipStream_t)HeTM_memStream[devId];
      //   }
      //   src0 += gran;
      //   dst0 += gran;
      //   src1 += gran;
      //   dst1 += gran;
      // }
      __atomic_add_fetch(&HeTM_stats_data.sizeCpyWSetHtD, cpywrsetBMAP, __ATOMIC_ACQ_REL);

      launchInterGPUConflDetectKernel(threadDataj, j, devId);
      launchInterGPUConflDetectKernel(threadDatak, devId, j);
    }
    countDevs++; // same thread
    if (countDevs == Config::GetInstance()->NbGPUs()) {
      lastBatch = batch;
      countDevs = 0;
    }
  }
}

static void waitGPUtoGPUconflictCompare()
{
  for (int j = 0; j < Config::GetInstance()->NbGPUs(); ++j)
  {
    Config::GetInstance()->SelDev(j);
    CUDA_CHECK_ERROR(hipDeviceSynchronize(), "waiting cross GPU validation");
    HETM_DEB_THRD_GPU("isInterGPUConflDone[%d] = 1\n", j);
    __atomic_store_n(&(HeTM_shared_data[j].isInterGPUConflDone), 1, __ATOMIC_RELEASE);
  }
}

static int launchInterGPUConflDetectKernel(HeTM_thread_s *threadData, int locGPU, int remGPU)
{
  // TODO: add early validation kernel
  // if (!doApply) { return 0; }

  // for (int j = 0; j < HETM_NB_DEVICES; ++j) {
  long nbGrans = HeTM_gshared_data.sizeMemPool / PR_LOCK_GRANULARITY;
  Config::GetInstance()->SelDev(locGPU);
  int nbThreadsX = 256;
  int bo = (nbGrans + nbThreadsX-1) / (nbThreadsX);

  // Memory region of the entry object
  // printf("dev = %i batchCount = %li\n", j, HeTM_shared_data[j].batchCount);
  // thread_local static HeTM_cmp_s checkTxCompressed_args;
  MemObj *m = HeTM_interGPUConflDetect->entryObj->GetMemObj(locGPU);
  HeTM_cmp_s *checkTxCompressed_args = (HeTM_cmp_s*)m->host;
  checkTxCompressed_args->knlArgs.devId = locGPU;
  checkTxCompressed_args->knlArgs.knlGlobal  = *(HeTM_get_global_arg(locGPU));
  checkTxCompressed_args->knlArgs.otherDevId = remGPU;
  checkTxCompressed_args->knlArgs.nbOfGPUs   = Config::GetInstance()->NbGPUs();
  checkTxCompressed_args->knlArgs.sizeWSet   = (int)nbGrans;
  checkTxCompressed_args->knlArgs.sizeRSet   = (int)HeTM_gshared_data.rsetLogSize;
  // checkTxCompressed_args->knlArgs.idCPUThr   = (int)threadData->id;
  checkTxCompressed_args->knlArgs.batchCount = (unsigned char) HeTM_gshared_data.batchCount;
  checkTxCompressed_args->clbkArgs = threadData; // TODO: pass some structure with the CUDA events for GPU<->GPU comparison

  HETM_DEB_THRD_GPU("\033[0;36m" "GPU %i HeTM_interGPUConflDetect (to check against GPU %i)" "\033[0m" ,
    locGPU, remGPU);
  
  // HeTM_interGPUConflDetect.select(&HeTM_interGPUConflDetect);
  // HeTM_interGPUConflDetect.setNbBlocks(bo, 1, 1);
  // HeTM_interGPUConflDetect.setThrsPerBlock(nbThreadsX, 1, 1);
  // HeTM_interGPUConflDetect.setDevice(locGPU);
  // HeTM_interGPUConflDetect.setStream(HeTM_memStream[locGPU]);
  // HeTM_interGPUConflDetect.setArgs(&checkTxCompressed_args);

  // // hipStreamSynchronize((hipStream_t)HeTM_memStream[locGPU]);
  // HeTM_interGPUConflDetect.run();

  m->CpyHtD(HeTM_memStream[locGPU]);
  HeTM_CPUGPUConflDetect->blocks  = (knlman_dim3_s){ .x = bo,         .y = 1, .z = 1 };
  HeTM_CPUGPUConflDetect->threads = (knlman_dim3_s){ .x = nbThreadsX, .y = 1, .z = 1 };
  CUDA_CHECK_ERROR(hipStreamSynchronize((hipStream_t)HeTM_memStream2[locGPU]), "");
  HeTM_CPUGPUConflDetect->Run(locGPU, HeTM_memStream[locGPU]);


  

  return 0;
}

void
mergeMatricesAndRunFVS(
  int nonBlock
) {
  // TODO: matrices use Unified Memory (1B * (NB_GPU+1*CPU)**2), 9B for 2 GPUs (is it worth the extra code?)
  HETM_DEB_THRD_GPU(" -0- GPU waits CPU side comparisons \n");
  if (!nonBlock)
    HeTM_sync_next_batch(); // wait CPU side comparisons
  mergeAllMatrices();

#ifdef HETM_DEB
  printMatrixPerGPU();
#endif

  // Get if any conflicts (compute feedback vertex set)
  graph G = fromSquareMat(Config::GetInstance()->NbGPUs()+1, (unsigned char*)HeTM_gshared_data.mat_confl_CPU_unif);
  BB_reset(G);
  BB_run();
  HETM_DEB_THRD_GPU(" -1- GPU notifies CPU of BB done \n");
  if (!nonBlock)
    HeTM_sync_BB(); // notify CPU to proceed
}
/* 
static void memman_cpy_to_cpu_fn(void *hostPtr, void *devPtr, size_t cpySize)
{
  CUDA_CPY_TO_HOST_ASYNC(hostPtr, devPtr, cpySize, (hipStream_t)HeTM_memStream[PR_curr_dev]);
}

static void memman_cpy_to_gpu_fn(void *hostPtr, void *devPtr, size_t cpySize)
{
  CUDA_CPY_TO_DEV_ASYNC(devPtr, hostPtr, cpySize, (hipStream_t)HeTM_memStream[PR_curr_dev]);
}
 */
void accumulateStatistics()
{
  // WAIT_ON_FLAG(isGetStatsDone); // TODO
  int anyAbort = 0;
  int nGPUs = Config::GetInstance()->NbGPUs();

  if (HeTM_gshared_data.isCPUEnabled && HeTM_gshared_data.isGPUEnabled)
  {
    int *sol = BB_getBestSolution();
    int *toRemove = BB_getFVS();
    const int CPUid = nGPUs;

    int *p = sol;
    while (*p != -1)
    {
      if (*p == CPUid)
        HeTM_stats_data.nbCommittedTxsCPU += lastRoundTXs[*p];
      else // TODO: transform this stat in an array
        HeTM_stats_data.nbCommittedTxsGPU += lastRoundTXs[*p];
      p++;
    }

    p = toRemove;
    while (*p != -1)
    {
      anyAbort = 1;
      if (*p == CPUid)
        HeTM_stats_data.nbDroppedTxsCPU += lastRoundTXs[*p];
      else // TODO: transform this stat in an array
        HeTM_stats_data.nbDroppedTxsGPU += lastRoundTXs[*p];
      p++;
    }
  }
  else
  {
    for (int j = 0; j < nGPUs; ++j)
      HeTM_stats_data.nbCommittedTxsGPU += lastRoundTXs[j];
  }

  HeTM_stats_data.nbBatches++;
  if (anyAbort) {
    HeTM_stats_data.nbBatchesFail++;
  } else {
    // printf("HeTM_stats_data.nbBatchesSuccess = %d\n", HeTM_stats_data.nbBatchesSuccess);
    HeTM_stats_data.nbBatchesSuccess++;
  }

  for (int j = 0; j < nGPUs; ++j)
  {
    PR_curr_dev = j;
    PR_checkpointAbortsCommits();
  }
}

void syncGPUdataset(void *args)
{
  HETM_DEB_THRD_GPU("Syncing dataset ...");

  NVTX_PUSH_RANGE("wait in GPU", NVTX_PROF_GPU_WAITS_NEXT_BATCH);
  // TODO: need to wait CPU validation
  mergeMatricesAndRunFVS(0);
  NVTX_POP_RANGE(); // NVTX_PROF_GPU_WAITS_NEXT_BATCH

  // printf("conf mat:\n");
  // for (int j = 0; j < HETM_NB_DEVICES+1; ++j) {
  //   for (int k = 0; k < HETM_NB_DEVICES+1; ++k) {
  //     printf("%d", (int) HeTM_gshared_data.mat_confl_CPU_unif[(HETM_NB_DEVICES+1)*j+k]);
  //   }
  //   printf("\n");
  // }
}

void waitGPUdataset(void *args)
{
  // HeTM_thread_s *threadData = (HeTM_thread_s*)args;

  for (int i = 0; i < Config::GetInstance()->NbGPUs(); ++i)
  {
    Config::GetInstance()->SelDev(i);
    
    // hipStreamSynchronize((hipStream_t)HeTM_memStream[i]);
    // hipStreamSynchronize((hipStream_t)HeTM_memStream2[i]);
    CUDA_CHECK_ERROR(hipDeviceSynchronize(), ""); // TODO: now I'm waiting this to complete
  }

  // TODO: now it is crashing
  // CUDA_EVENT_SYNCHRONIZE(threadData->cpyDatasetStartEvent);
  // CUDA_EVENT_SYNCHRONIZE(threadData->cpyDatasetStopEvent);

  // CUDA_EVENT_ELAPSED_TIME(&threadData->timeCpyDataset, threadData->cpyDatasetStartEvent,
  //   threadData->cpyDatasetStopEvent);
  // threadData->timeCpyDatasetSum += threadData->timeCpyDataset;
  __atomic_store_n(&isDatasetSyncDone, 1, __ATOMIC_RELEASE);
}

void getGPUPRSTMStats(void *argPtr)
{
  // TODO: this is empty!!!
  // HeTM_thread_s *threadData = (HeTM_thread_s*)argPtr;

  // PR_retrieveIO(&HeTM_pr_args);
  // threadData->curNbTxs = PR_nbCommits;
  // HeTM_stats_data.nbAbortsGPU += PR_nbAborts;
  // PR_resetStatistics(&HeTM_pr_args);
  __atomic_store_n(&isGetPRStatsDone, 1, __ATOMIC_RELEASE);
}

void getGPUStatistics(void *arg)
{
  int nGPUs = Config::GetInstance()->NbGPUs();
  long committedTxsCPUBatch = 0;
  long txsNonBlocking = 0;
  // long droppedTxsCPUBatch = 0; // TODO: in apply dataset;
  // int idGPUThread = HeTM_shared_data.nbCPUThreads; // the last one

  for (int i = 0; i < HeTM_gshared_data.nbCPUThreads; ++i) {
    committedTxsCPUBatch += __atomic_load_n(&(HeTM_shared_data[0].threadsInfo[i].curNbTxs), __ATOMIC_ACQUIRE);
    txsNonBlocking += __atomic_load_n(&(HeTM_shared_data[0].threadsInfo[i].curNbTxsNonBlocking), __ATOMIC_ACQUIRE);
    __atomic_store_n(&(HeTM_shared_data[0].threadsInfo[i].curNbTxs), 0, __ATOMIC_RELEASE);
    __atomic_store_n(&(HeTM_shared_data[0].threadsInfo[i].curNbTxsNonBlocking), 0, __ATOMIC_RELEASE);
  }

  lastRoundTXs[nGPUs] = committedTxsCPUBatch + txsNonBlocking;

  // TODO: not doing anythings
  choose_policy(0, arg); // choose the policy for the next batch

  HeTM_stats_data.nbTxsCPU += /*droppedTxsCPUBatch + */committedTxsCPUBatch;
  HeTM_stats_data.txsNonBlocking += txsNonBlocking; // assert == 0

  // TODO: add committed and/or dropped TXs (lastRoundTXs[nGPUs]) after resolving conflicts 
  // HeTM_stats_data.nbCommittedTxsCPU += committedTxsCPUBatch;
  // HeTM_stats_data.nbDroppedTxsCPU   += droppedTxsCPUBatch;

  PR_global_data_s *d;
  for (int j = 0; j < nGPUs; j++)
  {
    PR_curr_dev = j;
    d = &(PR_global[j]);
    HeTM_stats_data.nbTxsGPU += d->PR_nbCommitsSinceCheckpoint;
    HeTM_stats_data.nbTxsPerGPU[j] += d->PR_nbCommitsSinceCheckpoint;
    // printf("    dev%i: +%li TXs (%li total)\n", j, d->PR_nbCommitsSinceCheckpoint, HeTM_stats_data.nbTxsPerGPU[j]);
    HeTM_stats_data.nbAbortsGPU += d->PR_nbAbortsSinceCheckpoint;
    HeTM_stats_data.nbAbortsPerGPU[j] += d->PR_nbAbortsSinceCheckpoint;
    lastRoundTXs[j] = d->PR_nbCommitsSinceCheckpoint;
  }

  // printf(" >>> Update stats <<<\n");
  __atomic_store_n(&isGetStatsDone, 1, __ATOMIC_RELEASE);
}

void offloadResetGPUState(void*)
{
  HeTM_reset_GPU_state(roundCountAfterBatch); // flags/locks
  __atomic_store_n(&isGPUResetDone, 1, __ATOMIC_RELEASE);
}

void setCountAfterBatch()
{
  roundCountAfterBatch = *hetm_batchCount;
}

void runPrSTMCallback(int nbBlcks, int nbThrsPerBlck, void(*callback)(PR_globalKernelArgs), void* inPtr, int sizeIn, void* outPtr, int sizeOut)
{

  for (int j = 0; j < Config::GetInstance()->NbGPUs(); ++j)
  {
		pr_buffer_s inBuf, outBuf;

    // TODO: change PR-STM to use knlman
    Config::GetInstance(j)->SelDev(j);
		PR_curr_dev = j;

    PR_global_data_s *d = &(PR_global[PR_curr_dev]);
    
    d->PR_blockNum = nbBlcks;
    d->PR_threadNum = nbThrsPerBlck;

    inBuf.buf = inPtr;
    inBuf.size = sizeIn;
    outBuf.buf = outPtr;
    outBuf.size = sizeOut;
    PR_prepareIO(&HeTM_pr_args[j], inBuf, outBuf);

    PR_run(callback, &HeTM_pr_args[j]);

    // PR_i_cudaPrepare((&HeTM_pr_args[j]), bankTx);
    // PR_BEFORE_RUN_EXT((&HeTM_pr_args[j]));
    // // PR_i_run(pr_args);
    // bankTx<<<PR_blockNum,PR_threadNum,0,(hipStream_t)PR_getCurrentStream()>>>(HeTM_pr_args[j].dev);
    // PR_AFTER_RUN_EXT((&HeTM_pr_args[j]));

    // CUDA_CHECK_ERROR(hipDeviceSynchronize(), "");
  }
}
