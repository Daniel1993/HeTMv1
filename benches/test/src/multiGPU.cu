#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void someKernel(int devId)
{
  int cudaDevId;
  if (hipGetDevice(&cudaDevId) != hipSuccess) {
    printf("Cannot get device\n");
  }

  printf("hello from thread %i dev %i (cudaDev = %i)\n", threadIdx.x+blockDim.x*blockIdx.x, devId, cudaDevId);
  
}

int main(int argc, char **argv)
{
  int *dev0, *dev1; // TODO: array per device
  
  if (hipSetDevice(0) != hipSuccess) { printf("Cannot set dev 0\n"); }
  hipStream_t streamDev0;
  hipStreamCreate(&streamDev0);
  someKernel<<<1, 1, 0, streamDev0>>>(0);


  if (hipSetDevice(1) != hipSuccess) { printf("Cannot set dev 1\n"); }
  hipStream_t streamDev1;
  hipStreamCreate(&streamDev1);
  someKernel<<<1, 1, 0, streamDev1>>>(1);


  if (hipSetDevice(0) != hipSuccess) { printf("Cannot set dev 0\n"); }
  if (hipStreamSynchronize(streamDev0) != hipSuccess) { printf("Error in kernel dev 0\n"); }


  if (hipSetDevice(1) != hipSuccess) { printf("Cannot set dev 1\n"); }
  if (hipStreamSynchronize(streamDev1) != hipSuccess) { printf("Error in kernel dev 1\n"); }

  return 0;
}