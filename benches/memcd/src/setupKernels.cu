#include "hip/hip_runtime.h"
#include "hetm-log.h"

#include "setupKernels.cuh"
#include "cmp_kernels.cuh"
#include "bankKernel.cuh"
#include "bank.hpp"

#include "memman.hpp"

using namespace memman;
using namespace knlman;

KnlObj *HeTM_finalTxLog2;
KnlObj *HeTM_bankTx;
KnlObj *HeTM_memcdWriteTx;
KnlObj *HeTM_memcdReadTx;
MemObjOnDev HeTM_bankTxEntryObj;
MemObjOnDev HeTM_bankTxInput;
MemObjOnDev HeTM_memcdTx_input;
MemObjOnDev memcd_global_ts;

static void run_memcdReadTx(knlman_callback_params_s params);
static void run_memcdWriteTx(knlman_callback_params_s params);
static void run_finalTxLog2(knlman_callback_params_s params);

int HeTM_setup_memcdWriteTx(int nbBlocks, int nbThreads)
{
  PR_global_data_s *d;
  for (int j = 0; j < Config::GetInstance()->NbGPUs(); j++)
  {
    MemObjBuilder b;
    Config::GetInstance()->SelDev(j);
    PR_curr_dev = j;
    d = &(PR_global[PR_curr_dev]);
    d->PR_blockNum = nbBlocks;
    d->PR_threadNum = nbThreads;
    HeTM_memcdTx_input.AddMemObj(new MemObj(b
      .SetOptions(0)
      ->SetSize(sizeof(HeTM_memcdTx_input_s))
      ->AllocDevPtr()
      ->AllocHostPtr(),
      j
    ));
  }
  KnlObjBuilder b;
  HeTM_memcdWriteTx = new KnlObj(b
    .SetCallback(run_memcdWriteTx)
    ->SetEntryObj(&HeTM_memcdTx_input));
  return 0;
}

int HeTM_setup_memcdReadTx(int nbBlocks, int nbThreads)
{
  PR_global_data_s *d;
  for (int j = 0; j < HETM_NB_DEVICES; j++) {
    PR_curr_dev = j;
    d = &(PR_global[PR_curr_dev]);
    d->PR_blockNum = nbBlocks;
    d->PR_threadNum = nbThreads;
  }
  KnlObjBuilder b;
  HeTM_memcdReadTx = new KnlObj(b
    .SetCallback(run_memcdReadTx)
    ->SetEntryObj(&HeTM_memcdTx_input));

  // already set-up HeTM_memcdTx_input

  return 0;
}

int HeTM_bankTx_cpy_IO() // TODO: not used
{
  for (int j = 0; j < Config::GetInstance()->NbGPUs(); ++j)
  {
    Config::GetInstance()->SelDev(j);
    PR_curr_dev = j;
    pr_tx_args_s *pr_args = getPrSTMmetaData(j);
    PR_retrieveIO(pr_args);
  }
  return 0;
}

int HeTM_teardown_bankTx()
{
  delete HeTM_bankTx;
  return 0;
}

int HeTM_teardown_memcdWriteTx()
{
  delete HeTM_memcdWriteTx;
  return 0;
}

int HeTM_teardown_memcdReadTx()
{
  delete HeTM_memcdReadTx;
  return 0; 
}

int HeTM_setup_finalTxLog2()
{
  KnlObjBuilder b;
  HeTM_finalTxLog2 = new KnlObj(b
    .SetCallback(run_finalTxLog2));
  return 0;
}

int HeTM_teardown_finalTxLog2()
{
  // TODO: delete entryObj
  delete HeTM_finalTxLog2;
  return 0;
}

static void run_finalTxLog2(knlman_callback_params_s params)
{
  dim3 blocks(params.blocks.x, params.blocks.y, params.blocks.z);
  dim3 threads(params.threads.x, params.threads.y, params.threads.z);
  HeTM_knl_finalTxLog2_s *data = (HeTM_knl_finalTxLog2_s*)params.entryObj;

  /* Kernel Launch */
  HeTM_knl_finalTxLog2 <<< blocks, threads >>> (*data);

  // HeTM_knl_finalTxLog2<<<blocks, threads>>>(data->knlArgs);
}

static void run_memcdReadTx(knlman_callback_params_s params)
{
  HeTM_bankTx_s *data = (HeTM_bankTx_s*)params.entryObj; // TODO
  account_t *a = data->knlArgs.a;
  account_t *accounts = a;
  cuda_t *d = data->knlArgs.d;
  pr_buffer_s inBuf, outBuf;
  HeTM_memcdTx_input_s *input, *inputDev;

  // thread_local static unsigned short seed = 1234;

  for (int j = 0; j < HETM_NB_DEVICES; ++j)
  {
    Config::GetInstance()->SelDev(j);
    PR_curr_dev = j;

    CUDA_CHECK_ERROR(hipDeviceSynchronize(), ""); // sync the previous run

    // memman_ad_hoc_free(NULL); // empties the previous parameters
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(memcdReadTx), hipFuncCachePreferL1);

    if (a == NULL) {
      // This seems to swap the buffers if given a NULL array...
      accounts = d->dev_a;
      d->dev_a = d->dev_b;
      d->dev_b = accounts;
    }

    input = (HeTM_memcdTx_input_s*)HeTM_memcdTx_input.GetMemObj(j)->host;
    inputDev = (HeTM_memcdTx_input_s*)HeTM_memcdTx_input.GetMemObj(j)->dev;

    input->key      = d->dev_a;
    // TODO: /sizeof(...)
    input->extraKey = input->key + (d->memcd_nbSets*d->memcd_nbWays);
    input->val      = input->extraKey + 3*(d->memcd_nbSets*d->memcd_nbWays);
    input->extraVal = input->val + (d->memcd_nbSets*d->memcd_nbWays);
    input->ts_CPU   = input->extraVal + 7*(d->memcd_nbSets*d->memcd_nbWays);
    input->ts_GPU   = input->ts_CPU + (d->memcd_nbSets*d->memcd_nbWays);
    input->state    = input->ts_GPU + (d->memcd_nbSets*d->memcd_nbWays);
    input->setUsage = input->state + (d->memcd_nbSets*d->memcd_nbWays);
    input->nbSets   = d->num_sets;
    input->nbWays   = d->num_ways;
    input->input_keys = GPUInputBuffer[j];
    input->input_vals = GPUInputBuffer[j];
    input->output     = (memcd_get_output_t*)GPUoutputBuffer[j];

    input->curr_clock = (int*)memcd_global_ts.GetMemObj(j)->dev;

    HeTM_memcdTx_input.GetMemObj(j)->CpyHtD(HeTM_memStream2[j]);

    // TODO:
    // inputDev = (HeTM_memcdTx_input_s*)memman_ad_hoc_alloc(NULL, &input, sizeof(HeTM_memcdTx_input_s));
    // memman_ad_hoc_cpy(NULL);

    // TODO: change PR-STM to use knlman
    // PR_blockNum = params.blocks.x;
    // PR_threadNum = params.threads.x;
    inBuf.buf = (void*)inputDev;
    inBuf.size = sizeof(HeTM_memcdTx_input_s);
    outBuf.buf = NULL;
    outBuf.size = 0;
    pr_tx_args_s *pr_args = getPrSTMmetaData(j);
    PR_prepareIO(pr_args, inBuf, outBuf);
    PR_run(memcdReadTx, pr_args);
  }
}

static void run_memcdWriteTx(knlman_callback_params_s params)
{
  HeTM_bankTx_s *data = (HeTM_bankTx_s*)params.entryObj;
  account_t *a = data->knlArgs.a;
  account_t *accounts = a;
  cuda_t *d = data->knlArgs.d;
  pr_buffer_s inBuf, outBuf;
  HeTM_memcdTx_input_s *input, *inputDev;

  for (int j = 0; j < HETM_NB_DEVICES; ++j)
  {
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(memcdWriteTx), hipFuncCachePreferL1);

    if (a == NULL) {
      // This seems to swap the buffers if given a NULL array...
      accounts = d->dev_a;
      d->dev_a = d->dev_b;
      d->dev_b = accounts;
    }

    input = (HeTM_memcdTx_input_s*)HeTM_memcdTx_input.GetMemObj(j)->host;
    inputDev = (HeTM_memcdTx_input_s*)HeTM_memcdTx_input.GetMemObj(j)->dev;

    input->key   = d->dev_a;
    input->extraKey = input->key + (d->memcd_nbSets*d->memcd_nbWays);
    input->val      = input->extraKey + 3*(d->memcd_nbSets*d->memcd_nbWays);
    input->extraVal = input->val + (d->memcd_nbSets*d->memcd_nbWays);
    input->ts_CPU   = input->extraVal + 7*(d->memcd_nbSets*d->memcd_nbWays);
    input->ts_GPU   = input->ts_CPU + (d->memcd_nbSets*d->memcd_nbWays);
    input->state    = input->ts_GPU + (d->memcd_nbSets*d->memcd_nbWays);
    input->setUsage = input->state + (d->memcd_nbSets*d->memcd_nbWays);
    input->nbSets   = d->num_sets;
    input->nbWays   = d->num_ways;
    input->input_keys = GPUInputBuffer[j];
    input->input_vals = GPUInputBuffer[j];
    input->output     = (memcd_get_output_t*)GPUoutputBuffer[j];


    input->curr_clock = (int*)memcd_global_ts.GetMemObj(j)->dev;
    HeTM_memcdTx_input.GetMemObj(j)->CpyHtD(HeTM_memStream2[j]);

    // TODO: change PR-STM to use knlman
    // PR_blockNum = params.blocks.x;
    // PR_threadNum = params.threads.x;
    inBuf.buf = (void*)inputDev;
    inBuf.size = sizeof(HeTM_memcdTx_input_s);
    outBuf.buf = NULL;
    outBuf.size = 0;
    pr_tx_args_s *pr_args = getPrSTMmetaData(j);
    PR_prepareIO(pr_args, inBuf, outBuf);
    PR_run(memcdWriteTx, pr_args);
  }
}
