#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <iostream>
#include <memory>
#include <string>
#include <cstdio>
#include <cstdlib>

#include "cuda_util.h"

#define CONVERT_TO_GBPS(dataSize, timeMs) (((float)(dataSize)) / (((float)(timeMs) / 1000.0f) * (1024.0f*1024.0f*1024.0f)))

using namespace std;

int
main()
{
  int deviceCount;
  void **devMemDst;
  void *hostMemDst;
  void *hostMemDst2;
  void **devMemSrc;
  void *hostMemSrc;
  void *hostMemSrc2;
  const size_t COPY_SIZE = 1 * 1024*1024*1024; // 1GB
  hipEvent_t *cpyEv;
  hipStream_t s1;
  hipStream_t s2;
  hipStream_t s3;
  hipStream_t s4;
  hipStream_t s5;
  hipStream_t s6;
  float time_ms1, time_ms2;

  CUDA_CHECK_ERROR(hipGetDeviceCount(&deviceCount), "");

  printf("detected %d devices\n", deviceCount);
  CUDA_CHECK_ERROR(hipHostMalloc(&hostMemDst, COPY_SIZE), "");
  CUDA_CHECK_ERROR(hipHostMalloc(&hostMemDst2, COPY_SIZE), "");
  CUDA_CHECK_ERROR(hipHostMalloc(&hostMemSrc, COPY_SIZE), "");
  CUDA_CHECK_ERROR(hipHostMalloc(&hostMemSrc2, COPY_SIZE), "");
  devMemDst = (void**)malloc(deviceCount*sizeof(void*));
  devMemSrc = (void**)malloc(deviceCount*sizeof(void*));
  cpyEv = (hipEvent_t*)malloc(2*deviceCount*sizeof(hipEvent_t));
  CUDA_CHECK_ERROR(hipStreamCreate(&s1), "");
  CUDA_CHECK_ERROR(hipStreamCreate(&s2), "");
  CUDA_CHECK_ERROR(hipStreamCreate(&s3), "");
  CUDA_CHECK_ERROR(hipStreamCreate(&s4), "");
  CUDA_CHECK_ERROR(hipStreamCreate(&s5), "");
  CUDA_CHECK_ERROR(hipStreamCreate(&s6), "");
  for (int i = 0; i < deviceCount; ++i)
  {
    CUDA_CHECK_ERROR(hipSetDevice(i), "");

    CUDA_CHECK_ERROR(hipEventCreate(&(cpyEv[2*i])), "");
    CUDA_CHECK_ERROR(hipEventCreate(&(cpyEv[2*i+1])), "");
    CUDA_CHECK_ERROR(hipMalloc(&(devMemSrc[i]), COPY_SIZE), "");
    CUDA_CHECK_ERROR(hipMalloc(&(devMemDst[i]), COPY_SIZE), "");
  }

  // TODO: agnostic of the number of devices

  int canDev0AccessDev1, canDev1AccessDev0;
  CUDA_CHECK_ERROR(hipDeviceCanAccessPeer(&canDev0AccessDev1, 0, 1), "");
  CUDA_CHECK_ERROR(hipDeviceCanAccessPeer(&canDev1AccessDev0, 1, 0), "");
  if (canDev0AccessDev1) {
    printf("dev0 can access dev1 directly\n");
    CUDA_CHECK_ERROR(hipSetDevice(0), "");
    CUDA_CHECK_ERROR(hipDeviceEnablePeerAccess(/*peer_id*/1, 0), "");
  }
  if (canDev1AccessDev0) {
    printf("dev1 can access dev0 directly\n");
    CUDA_CHECK_ERROR(hipSetDevice(1), "");
    CUDA_CHECK_ERROR(hipDeviceEnablePeerAccess(/*peer_id*/0, 0), "");
  }

  CUDA_CHECK_ERROR(hipSetDevice(0), "");
  
  CUDA_CHECK_ERROR(hipEventRecord(cpyEv[0], s1), "");
  CUDA_CPY_PtP_ASYNC(devMemDst[0], 0, devMemSrc[1], 1, COPY_SIZE, s1);
  CUDA_CHECK_ERROR(hipEventRecord(cpyEv[1], s1), "");
  CUDA_CHECK_ERROR(hipStreamSynchronize(s1), "");
  CUDA_CHECK_ERROR(hipEventSynchronize(cpyEv[0]), "");
  CUDA_CHECK_ERROR(hipEventSynchronize(cpyEv[1]), "");
  CUDA_CHECK_ERROR(hipEventElapsedTime(&time_ms1, cpyEv[0], cpyEv[1]), "");
  printf("CPY PtP dev1->dev0 took %f ms (%f GB/s)\n", time_ms1, CONVERT_TO_GBPS(COPY_SIZE, time_ms1));

  CUDA_CHECK_ERROR(hipSetDevice(1), "");

  CUDA_CHECK_ERROR(hipEventRecord(cpyEv[0], s1), "");
  CUDA_CPY_PtP_ASYNC(devMemDst[1], 1, devMemSrc[0], 0, COPY_SIZE, s1);
  CUDA_CHECK_ERROR(hipEventRecord(cpyEv[1], s1), "");
  CUDA_CHECK_ERROR(hipStreamSynchronize(s1), "");
  CUDA_CHECK_ERROR(hipEventSynchronize(cpyEv[0]), "");
  CUDA_CHECK_ERROR(hipEventSynchronize(cpyEv[1]), "");
  CUDA_CHECK_ERROR(hipEventElapsedTime(&time_ms1, cpyEv[0], cpyEv[1]), "");
  printf("CPY PtP dev0->dev1 took %f ms (%f GB/s)\n", time_ms1, CONVERT_TO_GBPS(COPY_SIZE, time_ms1));

  CUDA_CHECK_ERROR(hipSetDevice(0), "");

  CUDA_CHECK_ERROR(hipEventRecord(cpyEv[0], s1), "");
  CUDA_CPY_TO_DEV_ASYNC(devMemDst[0], hostMemSrc, COPY_SIZE, s1);
  CUDA_CHECK_ERROR(hipEventRecord(cpyEv[1], s1), "");
  CUDA_CHECK_ERROR(hipStreamSynchronize(s1), "");
  CUDA_CHECK_ERROR(hipEventSynchronize(cpyEv[0]), "");
  CUDA_CHECK_ERROR(hipEventSynchronize(cpyEv[1]), "");
  CUDA_CHECK_ERROR(hipEventElapsedTime(&time_ms1, cpyEv[0], cpyEv[1]), "");
  printf("CPY HtD host->dev0 took %f ms (%f GB/s)\n", time_ms1, CONVERT_TO_GBPS(COPY_SIZE, time_ms1));

  CUDA_CHECK_ERROR(hipSetDevice(1), "");

  CUDA_CHECK_ERROR(hipEventRecord(cpyEv[0], s1), "");
  CUDA_CPY_TO_DEV_ASYNC(devMemDst[1], hostMemSrc, COPY_SIZE, s1);
  CUDA_CHECK_ERROR(hipEventRecord(cpyEv[1], s1), "");
  CUDA_CHECK_ERROR(hipStreamSynchronize(s1), "");
  CUDA_CHECK_ERROR(hipEventSynchronize(cpyEv[0]), "");
  CUDA_CHECK_ERROR(hipEventSynchronize(cpyEv[1]), "");
  CUDA_CHECK_ERROR(hipEventElapsedTime(&time_ms1, cpyEv[0], cpyEv[1]), "");
  printf("CPY HtD host->dev1 took %f ms (%f GB/s)\n", time_ms1, CONVERT_TO_GBPS(COPY_SIZE, time_ms1));

  CUDA_CHECK_ERROR(hipSetDevice(0), "");

  CUDA_CHECK_ERROR(hipEventRecord(cpyEv[0], s1), "");
  CUDA_CPY_TO_HOST_ASYNC(hostMemDst, devMemSrc[0], COPY_SIZE, s1);
  CUDA_CHECK_ERROR(hipEventRecord(cpyEv[1], s1), "");
  CUDA_CHECK_ERROR(hipStreamSynchronize(s1), "");
  CUDA_CHECK_ERROR(hipEventSynchronize(cpyEv[0]), "");
  CUDA_CHECK_ERROR(hipEventSynchronize(cpyEv[1]), "");
  CUDA_CHECK_ERROR(hipEventElapsedTime(&time_ms1, cpyEv[0], cpyEv[1]), "");
  printf("CPY DtH dev0->host took %f ms (%f GB/s)\n", time_ms1, CONVERT_TO_GBPS(COPY_SIZE, time_ms1));

  CUDA_CHECK_ERROR(hipSetDevice(1), "");

  CUDA_CHECK_ERROR(hipEventRecord(cpyEv[0], s1), "");
  CUDA_CPY_TO_HOST_ASYNC(hostMemDst, devMemSrc[1], COPY_SIZE, s1);
  CUDA_CHECK_ERROR(hipEventRecord(cpyEv[1], s1), "");
  CUDA_CHECK_ERROR(hipStreamSynchronize(s1), "");
  CUDA_CHECK_ERROR(hipEventSynchronize(cpyEv[0]), "");
  CUDA_CHECK_ERROR(hipEventSynchronize(cpyEv[1]), "");
  CUDA_CHECK_ERROR(hipEventElapsedTime(&time_ms1, cpyEv[0], cpyEv[1]), "");
  printf("CPY DtH dev1->host took %f ms (%f GB/s)\n", time_ms1, CONVERT_TO_GBPS(COPY_SIZE, time_ms1));

  // CUDA_CHECK_ERROR(hipSetDevice(1), "");

  CUDA_CHECK_ERROR(hipSetDevice(0), "");
  CUDA_CHECK_ERROR(hipEventRecord(cpyEv[0], s1), "");
  CUDA_CPY_PtP_ASYNC(devMemDst[0], 0, devMemSrc[1], 1, COPY_SIZE, s1);
  CUDA_CHECK_ERROR(hipEventRecord(cpyEv[1], s1), "");
  CUDA_CHECK_ERROR(hipSetDevice(1), "");
  // CUDA_CHECK_ERROR(hipEventRecord(cpyEv[2], s2), "");
  CUDA_CPY_PtP_ASYNC(devMemDst[1], 1, devMemSrc[0], 0, COPY_SIZE, s2);
  // CUDA_CHECK_ERROR(hipEventRecord(cpyEv[3], s2), "");
  CUDA_CHECK_ERROR(hipStreamSynchronize(s1), "");
  CUDA_CHECK_ERROR(hipEventSynchronize(cpyEv[0]), "");
  CUDA_CHECK_ERROR(hipEventSynchronize(cpyEv[1]), "");
  CUDA_CHECK_ERROR(hipStreamSynchronize(s2), "");
  CUDA_CHECK_ERROR(hipEventSynchronize(cpyEv[2]), "");
  CUDA_CHECK_ERROR(hipEventSynchronize(cpyEv[3]), "");
  CUDA_CHECK_ERROR(hipEventElapsedTime(&time_ms1, cpyEv[0], cpyEv[1]), "");
  // CUDA_CHECK_ERROR(hipEventElapsedTime(&time_ms2, cpyEv[2], cpyEv[3]), "");
  printf("CPY concurrent PtP dev0->dev1 took %f ms (%f GB/s)\n", time_ms1, CONVERT_TO_GBPS(COPY_SIZE, time_ms1));
  // printf("CPY concurrent PtP dev1->dev0 took %f ms (%f GB/s)\n", time_ms2, CONVERT_TO_GBPS(COPY_SIZE, time_ms2));


  CUDA_CHECK_ERROR(hipSetDevice(0), "");
  CUDA_CPY_TO_HOST_ASYNC(hostMemDst, devMemSrc[0], COPY_SIZE, s1);
  CUDA_CHECK_ERROR(hipSetDevice(1), "");
  CUDA_CPY_TO_HOST_ASYNC(hostMemDst, devMemSrc[1], COPY_SIZE, s2);
  CUDA_CHECK_ERROR(hipStreamSynchronize(s1), "");
  CUDA_CHECK_ERROR(hipStreamSynchronize(s2), "");

  CUDA_CHECK_ERROR(hipSetDevice(0), "");
  CUDA_CPY_TO_DEV_ASYNC(devMemDst[0], hostMemSrc, COPY_SIZE, s1);
  CUDA_CHECK_ERROR(hipSetDevice(1), "");
  CUDA_CPY_TO_DEV_ASYNC(devMemDst[1], hostMemSrc, COPY_SIZE, s2);
  CUDA_CHECK_ERROR(hipStreamSynchronize(s1), "");
  CUDA_CHECK_ERROR(hipStreamSynchronize(s2), "");

  CUDA_CHECK_ERROR(hipSetDevice(0), "");
  CUDA_CPY_TO_DEV_ASYNC(devMemDst[0], hostMemSrc, COPY_SIZE, s1);
  CUDA_CHECK_ERROR(hipSetDevice(1), "");
  CUDA_CPY_TO_HOST_ASYNC(hostMemDst, devMemSrc[1], COPY_SIZE, s2);
  CUDA_CHECK_ERROR(hipStreamSynchronize(s1), "");
  CUDA_CHECK_ERROR(hipStreamSynchronize(s2), "");

  CUDA_CHECK_ERROR(hipSetDevice(0), "");
  CUDA_CPY_TO_HOST_ASYNC(hostMemDst, devMemSrc[0], COPY_SIZE, s1);
  CUDA_CHECK_ERROR(hipSetDevice(1), "");
  CUDA_CPY_TO_DEV_ASYNC(devMemDst[1], hostMemSrc, COPY_SIZE, s2);
  CUDA_CHECK_ERROR(hipStreamSynchronize(s1), "");
  CUDA_CHECK_ERROR(hipStreamSynchronize(s2), "");

  CUDA_CHECK_ERROR(hipSetDevice(0), "");
  CUDA_CPY_TO_DEV_ASYNC(devMemDst[0], hostMemSrc, COPY_SIZE, s1);
  CUDA_CPY_TO_HOST_ASYNC(hostMemDst, devMemSrc[0], COPY_SIZE, s2);
  CUDA_CHECK_ERROR(hipStreamSynchronize(s1), "");
  CUDA_CHECK_ERROR(hipStreamSynchronize(s2), "");

  CUDA_CHECK_ERROR(hipSetDevice(1), "");
  CUDA_CPY_TO_DEV_ASYNC(devMemDst[1], hostMemSrc, COPY_SIZE, s3);
  CUDA_CPY_TO_HOST_ASYNC(hostMemDst, devMemSrc[1], COPY_SIZE, s4);
  CUDA_CHECK_ERROR(hipStreamSynchronize(s3), "");
  CUDA_CHECK_ERROR(hipStreamSynchronize(s4), "");

  CUDA_CHECK_ERROR(hipSetDevice(0), "");
  CUDA_CPY_TO_DEV_ASYNC(devMemDst[0], hostMemSrc, COPY_SIZE, s1);
  CUDA_CPY_TO_HOST_ASYNC(hostMemDst, devMemSrc[0], COPY_SIZE, s2);
  CUDA_CHECK_ERROR(hipSetDevice(1), "");
  CUDA_CPY_TO_DEV_ASYNC(devMemDst[1], hostMemSrc, COPY_SIZE, s3);
  CUDA_CPY_TO_HOST_ASYNC(hostMemDst, devMemSrc[1], COPY_SIZE, s4);
  CUDA_CHECK_ERROR(hipStreamSynchronize(s1), "");
  CUDA_CHECK_ERROR(hipStreamSynchronize(s2), "");
  CUDA_CHECK_ERROR(hipStreamSynchronize(s3), "");
  CUDA_CHECK_ERROR(hipStreamSynchronize(s4), "");

  CUDA_CHECK_ERROR(hipSetDevice(0), "");
  CUDA_CPY_TO_DEV_ASYNC(devMemDst[0], hostMemSrc, COPY_SIZE, s1);
  CUDA_CPY_TO_HOST_ASYNC(hostMemDst, devMemSrc[0], COPY_SIZE, s2);
  CUDA_CHECK_ERROR(hipSetDevice(1), "");
  CUDA_CPY_TO_DEV_ASYNC(devMemDst[1], hostMemSrc2, COPY_SIZE, s3);
  CUDA_CPY_TO_HOST_ASYNC(hostMemDst2, devMemSrc[1], COPY_SIZE, s4);
  CUDA_CHECK_ERROR(hipStreamSynchronize(s1), "");
  CUDA_CHECK_ERROR(hipStreamSynchronize(s2), "");
  CUDA_CHECK_ERROR(hipStreamSynchronize(s3), "");
  CUDA_CHECK_ERROR(hipStreamSynchronize(s4), "");

  CUDA_CHECK_ERROR(hipSetDevice(0), "");
  CUDA_CPY_TO_DEV_ASYNC(devMemDst[0], hostMemSrc, COPY_SIZE, s1);
  CUDA_CPY_TO_HOST_ASYNC(hostMemDst, devMemSrc[0], COPY_SIZE, s2);
  CUDA_CHECK_ERROR(hipSetDevice(1), "");
  CUDA_CPY_TO_DEV_ASYNC(devMemDst[1], hostMemSrc, COPY_SIZE, s3);
  CUDA_CPY_TO_HOST_ASYNC(hostMemDst, devMemSrc[1], COPY_SIZE, s4);
  CUDA_CPY_PtP_ASYNC(devMemDst[0], 0, devMemSrc[1], 1, COPY_SIZE, s5);
  CUDA_CPY_PtP_ASYNC(devMemDst[1], 1, devMemSrc[0], 0, COPY_SIZE, s6);
  CUDA_CHECK_ERROR(hipStreamSynchronize(s1), "");
  CUDA_CHECK_ERROR(hipStreamSynchronize(s2), "");
  CUDA_CHECK_ERROR(hipStreamSynchronize(s3), "");
  CUDA_CHECK_ERROR(hipStreamSynchronize(s4), "");
  CUDA_CHECK_ERROR(hipStreamSynchronize(s5), "");
  CUDA_CHECK_ERROR(hipStreamSynchronize(s6), "");

  for (int i = 0; i < deviceCount; ++i)
  {
    hipEventDestroy(cpyEv[i]);
    hipEventDestroy(cpyEv[i+1]);
    hipFree(devMemSrc[i]);
    hipFree(devMemDst[i]);
  }
  free(devMemSrc);
  free(devMemDst);
  hipHostFree(hostMemSrc);
  hipHostFree(hostMemSrc2);
  hipHostFree(hostMemDst);
  hipHostFree(hostMemDst2);

  return 0;
}

